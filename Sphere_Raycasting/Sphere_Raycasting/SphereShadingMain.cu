#include "hip/hip_runtime.h"
﻿
#include <stdio.h>

#include "Dependencies/GL/glew.h"
#include "Dependencies/GL/freeglut.h"

#include "hip/hip_runtime.h"
#include ""
#include "cuda_gl_interop.h"

#include "Dependencies/Helpers/hip/hip_runtime_api.h"
#include "Dependencies/Helpers/helper_gl.h"
#include "Dependencies/Helpers/helper_timer.h"

//#include "SphereShadingCPU.h"
#include "SphereShading.cuh"
#include "Scene/SceneCreator.h"

#define REFRESH_DELAY 10  // ms
#define SPHERE_COUNT 1000
#define LIGHT_COUNT 10

// OpenGL PBO and texture "names"
GLuint gl_PBO, gl_Tex, gl_Shader;
struct hipGraphicsResource* cuda_pbo_resource;  // handles OpenGL-CUDA exchange

// Source image on the host side
uchar4* h_Src = NULL;

// Destination image on the GPU side
uchar4* d_dst = NULL;

// Check whether frames should be rendered by CPU or GPU
bool g_runcpu = false;

// Original image width and height
int imageW = 1600, imageH = 1000;

// Thread count per dimension
const int tx = 16, ty = 16;

// Starting position and scale
double xOff = -0.5;
double yOff = 0.0;
double scale = 3.2;

// Starting stationary position and scale motion
double xdOff = 0.0;
double ydOff = 0.0;
double dscale = 1.0;

// Starting color multipliers and random seed
int colorSeed = 0;
uchar4 colors;

int numSMs = 0;   // number of multiprocessors
int version = 1;  // Compute Capability

s_scene* h_scene;
s_scene d_scene_allocationhelper;
s_scene* d_scene;

float3 moveLEFT = { 1, 0, 0 };
float3 moveBACK = { 0, 0, 1 };
bool cameraRotationMode = false;
int ox, oy;
float angle_x = 0, angle_y = 0;
float start_angle_x, start_angle_y;

static int DisplaySize() {
    return imageW * imageH;
}

// Timer ID
StopWatchInterface* hTimer = NULL;
StopWatchInterface* globalTimer = NULL;

int frameCount, fpsCount, fpsLimit = 15;
float deltaTime = 0;
float lastFrameTime = 0;
float currentFrameTime = 0;
float framesPerSecond;

float copyToDeviceTime;
float calculationTime;
float copyToHostTime;
float timeFor60frames;
int frameiterator = 0;

#ifndef MAX
#define MAX(a, b) ((a > b) ? a : b)
#endif
#define BUFFER_DATA(i) ((char *)0 + i)

void computeFPS() {
    frameCount++;
    fpsCount++;
    if (fpsCount == fpsLimit) {
        char fps[256];
        float ifps = 1.f / (sdkGetAverageTimerValue(&hTimer) / 1000.f);
        framesPerSecond = ifps;
        sprintf(fps, "<CUDA %s Set> %3.1f fps, %5.8f render time",
            "Sphere Shading", ifps, deltaTime * 0.001f);
        glutSetWindowTitle(fps);
        fpsCount = 0;

        fpsLimit = (int)MAX(1.f, (float)ifps);
        sdkResetTimer(&hTimer);
    }
}

// gl_Shader for displaying floating-point texture
static const char* shader_code =
"!!ARBfp1.0\n"
"TEX result.color, fragment.texcoord, texture[0], 2D; \n"
"END";
GLuint compileASMShader(GLenum program_type, const char* code) {
    GLuint program_id;
    glGenProgramsARB(1, &program_id);
    glBindProgramARB(program_type, program_id);
    glProgramStringARB(program_type, GL_PROGRAM_FORMAT_ASCII_ARB,
        (GLsizei)strlen(code), (GLubyte*)code);

    GLint error_pos;
    glGetIntegerv(GL_PROGRAM_ERROR_POSITION_ARB, &error_pos);

    if (error_pos != -1) {
        const GLubyte* error_string;
        error_string = glGetString(GL_PROGRAM_ERROR_STRING_ARB);
        fprintf(stderr, "Program error at position: %d\n%s\n", (int)error_pos,
            error_string);
        return 0;
    }

    return program_id;
}

void initOpenGLBuffers(int w, int h) {
    // delete old buffers
    if (h_Src) {
        free(h_Src);
        h_Src = 0;
    }

    if (gl_Tex) {
        glDeleteTextures(1, &gl_Tex);
        gl_Tex = 0;
    }

    if (gl_PBO) {
        // DEPRECATED: checkCudaErrors(cudaGLUnregisterBufferObject(gl_PBO));
        hipGraphicsUnregisterResource(cuda_pbo_resource);
        glDeleteBuffers(1, &gl_PBO);
        gl_PBO = 0;
    }

    // allocate new buffers
    h_Src = (uchar4*)malloc(w * h * 4);

    printf("Creating GL texture...\n");
    glEnable(GL_TEXTURE_2D);
    glGenTextures(1, &gl_Tex);
    glBindTexture(GL_TEXTURE_2D, gl_Tex);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA8, w, h, 0, GL_RGBA, GL_UNSIGNED_BYTE,
        h_Src);
    printf("Texture created.\n");

    printf("Creating PBO...\n");
    glGenBuffers(1, &gl_PBO);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, gl_PBO);
    glBufferData(GL_PIXEL_UNPACK_BUFFER_ARB, w * h * 4, h_Src, GL_STREAM_COPY);
    // While a PBO is registered to CUDA, it can't be used
    // as the destination for OpenGL drawing calls.
    // But in our particular case OpenGL is only used
    // to display the content of the PBO, specified by CUDA kernels,
    // so we need to register/unregister it only once.

    // DEPRECATED: checkCudaErrors( cudaGLRegisterBufferObject(gl_PBO) );
    checkCudaErrors(hipGraphicsGLRegisterBuffer(
        &cuda_pbo_resource, gl_PBO, cudaGraphicsMapFlagsWriteDiscard));
    printf("PBO created.\n");

    checkCudaErrors(hipMalloc(&d_dst, (w * h * 4) * sizeof(uchar4)));

    // load shader program
    gl_Shader = compileASMShader(GL_FRAGMENT_PROGRAM_ARB, shader_code);
}

static void keyboard(unsigned char key, int /*x*/, int /*y*/) {

    s_camera* cam = &h_scene->camera;

	switch (key) {
	case 'w':
        cam->position += -deltaTime * cam->speed * moveBACK;
		break;
	case 's':
        cam->position += deltaTime * cam->speed * moveBACK;
		break;
	case 'a':
        cam->position += -deltaTime * cam->speed * moveLEFT;
		break;
	case 'd':
        cam->position += deltaTime * cam->speed * moveLEFT;
		break;
    case 'g':
        g_runcpu = true;
        break;
    case 'G':
        g_runcpu = false;
        break;
	}

    glutPostRedisplay();
}

void clickFunc(int button, int state, int x, int y) {
    //if (button == GLUT_LEFT_BUTTON) {
    //    if (state == GLUT_DOWN) {
    //        cameraRotationMode = true;
    //        printf("leftclick\n");
    //    }
    //    else {
    //        cameraRotationMode = false;
    //        printf("rightclick\n");
    //    }
    //}

    if (state == GLUT_DOWN) {
        ox = x;
        oy = y;
        start_angle_x = angle_x;
        start_angle_y = angle_y;
    }

    glutPostRedisplay();
}

void motionFunc(int x, int y) {
    
    //angle_x = start_angle_x + ((float)(x - ox) / 300.0f);
    //angle_y = start_angle_y + ((float)(y - oy) / 300.0f);

    //float yaw = deltaTime * viewer.speed * angle_x;
    //float pitch = deltaTime * viewer.speed * angle_y;

    ////if (pitch > 89.0f) {
    ////    pitch = 89.0f;
    ////}
    ////else if (pitch < -89.0f) {
    ////    pitch = -89.0f;
    ////}

    ////viewer.forward.

    //vec3 forward = viewer.forward;

    //vec3 newForward = vec3(
    //    forward.x() - sin(yaw) * cos(pitch),
    //    forward.x() - sin(pitch),
    //    forward.x() - cos(yaw) * cos(pitch)
    //);

    //vec3 right = viewer.right;

    //vec3 newRight = vec3(
    //    right.x() - cos(yaw),
    //    right.y(),
    //    right.z() + sin(yaw)
    //);

    //viewer.setForward(unit_vector(newForward));
    //viewer.setRight(unit_vector(newRight));
    //viewer.setUp(unit_vector(cross(newForward, newRight)));

    ////printf("new UP %3.1f, %3.1f, %3.1f\n", viewer.front.x(), viewer.front.y(), viewer.front.z());

    //ox = x;
    //oy = y;

    glutPostRedisplay();
}

static void reshape(int w, int h) {
    glViewport(0, 0, w, h);

    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();

    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    glOrtho(0.0, 1.0, 0.0, 1.0, 0.0, 1.0);

    if (w != 0 && h != 0)  // Do not call when window is minimized that is when
                           // width && height == 0
        initOpenGLBuffers(w, h);

    imageW = w;
    imageH = h;
    set_resolution(h_scene->camera, imageH, imageW);
    //look_at(h_scene->camera, make_float3(0, 0, -1));

    glutPostRedisplay();
}

static void cleanup() {
    if (h_Src) {
        free(h_Src);
        h_Src = 0;
    }

    delete h_scene->spheres.center.x;
    delete h_scene->spheres.center.y;
    delete h_scene->spheres.center.z;
    delete h_scene->spheres.center.angle;
    delete h_scene->spheres.colors.x;
    delete h_scene->spheres.colors.y;
    delete h_scene->spheres.colors.z;
    delete h_scene->spheres.ka;
    delete h_scene->spheres.kd;
    delete h_scene->spheres.ks;
    delete h_scene->spheres.radius;
    delete h_scene->lights.center.x;
    delete h_scene->lights.center.y;
    delete h_scene->lights.center.z;
    delete h_scene->lights.center.angle;
    delete h_scene->lights.colors.x;
    delete h_scene->lights.colors.y;
    delete h_scene->lights.colors.z;
    delete h_scene;

    sdkStopTimer(&hTimer);
    sdkDeleteTimer(&hTimer);

    sdkStopTimer(&globalTimer);
    sdkDeleteTimer(&globalTimer);

    // DEPRECATED: checkCudaErrors(cudaGLUnregisterBufferObject(gl_PBO));
    checkCudaErrors(hipGraphicsUnregisterResource(cuda_pbo_resource));
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, 0);

    glDeleteBuffers(1, &gl_PBO);
    glDeleteTextures(1, &gl_Tex);
    glDeleteProgramsARB(1, &gl_Shader);

    hipFree(d_dst);

    hipFree(d_scene_allocationhelper.spheres.center.x);
    hipFree(d_scene_allocationhelper.spheres.center.y);
    hipFree(d_scene_allocationhelper.spheres.center.z);
    hipFree(d_scene_allocationhelper.spheres.center.angle);
    hipFree(d_scene_allocationhelper.spheres.colors.x);
    hipFree(d_scene_allocationhelper.spheres.colors.y);
    hipFree(d_scene_allocationhelper.spheres.colors.z);
    hipFree(d_scene_allocationhelper.spheres.ka);
    hipFree(d_scene_allocationhelper.spheres.kd);
    hipFree(d_scene_allocationhelper.spheres.ks);
    hipFree(d_scene_allocationhelper.spheres.radius);
    hipFree(d_scene_allocationhelper.lights.center.angle);
    hipFree(d_scene_allocationhelper.lights.center.x);
    hipFree(d_scene_allocationhelper.lights.center.y);
    hipFree(d_scene_allocationhelper.lights.center.z);
    hipFree(d_scene_allocationhelper.lights.colors.x);
    hipFree(d_scene_allocationhelper.lights.colors.y);
    hipFree(d_scene_allocationhelper.lights.colors.z);
    hipFree(d_scene);
}

static void copyHostToDevice_positions(s_positions& dst, s_positions& src, int count) {
    checkCudaErrors(hipMemcpy(dst.x, src.x, sizeof(float) * count, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(dst.y, src.y, sizeof(float) * count, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(dst.z, src.z, sizeof(float) * count, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(dst.angle, src.angle, sizeof(float) * count, hipMemcpyHostToDevice));
}
static void copyHostToDevice_colors(s_colors& dst, s_colors& src, int count) {
    checkCudaErrors(hipMemcpy(dst.x, src.x, sizeof(float) * count, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(dst.y, src.y, sizeof(float) * count, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(dst.z, src.z, sizeof(float) * count, hipMemcpyHostToDevice));
}
static void copyHostToDevice_spheres(s_spheres& dst, s_spheres& src) {
    int count = src.count;
    copyHostToDevice_positions(dst.center, src.center, count);
    checkCudaErrors(hipMemcpy(dst.radius, src.radius, sizeof(float) * count, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(dst.ka, src.ka, sizeof(float) * count, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(dst.kd, src.kd, sizeof(float) * count, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(dst.ks, src.ks, sizeof(float) * count, hipMemcpyHostToDevice));
    copyHostToDevice_colors(dst.colors, src.colors, count);
}
static void copyHostToDevice_lights(s_lights& dst, s_lights& src) {
    int count = src.count;
    copyHostToDevice_positions(dst.center, src.center, count);
    copyHostToDevice_colors(dst.colors, src.colors, count);
}
static void copyHostToDevice_float(float& dst, float& src) {
    checkCudaErrors(hipMemcpy(&dst, &src, sizeof(float), hipMemcpyHostToDevice));
}
static void copyHostToDevice_float3(float3& dst, float3& src) {
    checkCudaErrors(hipMemcpy(&dst.x, &src.x, sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(&dst.y, &src.y, sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(&dst.z, &src.z, sizeof(float), hipMemcpyHostToDevice));
}

static void copyHostMemoryToDevice(bool copySceneElements) {
    checkCudaErrors(hipMemcpy(d_scene, h_scene, sizeof(s_scene), hipMemcpyHostToDevice));
    if (copySceneElements == true) {
        copyHostToDevice_spheres(d_scene_allocationhelper.spheres, h_scene->spheres);
        copyHostToDevice_lights(d_scene_allocationhelper.lights, h_scene->lights);
    }

    checkCudaErrors(hipMemcpy(&d_scene->spheres.center.x, &d_scene_allocationhelper.spheres.center.x, sizeof(float*), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(&d_scene->spheres.center.y, &d_scene_allocationhelper.spheres.center.y, sizeof(float*), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(&d_scene->spheres.center.z, &d_scene_allocationhelper.spheres.center.z, sizeof(float*), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(&d_scene->spheres.center.angle, &d_scene_allocationhelper.spheres.center.angle, sizeof(float*), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(&d_scene->spheres.radius, &d_scene_allocationhelper.spheres.radius, sizeof(float*), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(&d_scene->spheres.ka, &d_scene_allocationhelper.spheres.ka, sizeof(float*), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(&d_scene->spheres.kd, &d_scene_allocationhelper.spheres.kd, sizeof(float*), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(&d_scene->spheres.ks, &d_scene_allocationhelper.spheres.ks, sizeof(float*), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(&d_scene->spheres.colors.x, &d_scene_allocationhelper.spheres.colors.x, sizeof(float*), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(&d_scene->spheres.colors.y, &d_scene_allocationhelper.spheres.colors.y, sizeof(float*), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(&d_scene->spheres.colors.z, &d_scene_allocationhelper.spheres.colors.z, sizeof(float*), hipMemcpyHostToDevice));

    checkCudaErrors(hipMemcpy(&d_scene->lights.center.x, &d_scene_allocationhelper.lights.center.x, sizeof(float*), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(&d_scene->lights.center.y, &d_scene_allocationhelper.lights.center.y, sizeof(float*), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(&d_scene->lights.center.z, &d_scene_allocationhelper.lights.center.z, sizeof(float*), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(&d_scene->lights.center.angle, &d_scene_allocationhelper.lights.center.angle, sizeof(float*), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(&d_scene->lights.colors.x, &d_scene_allocationhelper.lights.colors.x, sizeof(float*), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(&d_scene->lights.colors.y, &d_scene_allocationhelper.lights.colors.y, sizeof(float*), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(&d_scene->lights.colors.z, &d_scene_allocationhelper.lights.colors.z, sizeof(float*), hipMemcpyHostToDevice));
}

static void copyDeviceToHost() {
    checkCudaErrors(hipMemcpy(h_Src, d_dst, sizeof(uchar4) * imageH * imageW, hipMemcpyDeviceToHost));
}

void renderImage(bool runcpu) {
    if (runcpu) {
        //int startPass = pass;
        float xs, ys;
        xs = ys = 0;
		sdkResetTimer(&hTimer);

		checkCudaErrors(hipGraphicsMapResources(1, &cuda_pbo_resource, 0));
		size_t num_bytes;
		checkCudaErrors(hipGraphicsResourceGetMappedPointer(
			(void**)&d_dst, &num_bytes, cuda_pbo_resource));

		// Get the pixel scale and offset
		double s = scale / (double)imageW;
		double x = (xs - (double)imageW * 0.5f) * s + xOff;
		double y = (ys - (double)imageH * 0.5f) * s + yOff;

        // Run the mandelbrot generator
        renderImageCPU(h_Src, imageW, imageH, *h_scene);
        
        // Use the adaptive sampling version when animating.

        checkCudaErrors(hipMemcpy(d_dst, h_Src, imageW * imageH * sizeof(uchar4),
            hipMemcpyHostToDevice));

        checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_pbo_resource, 0));

        printf("CPU = %5.8f\n", 0.001f * sdkGetTimerValue(&hTimer));
    }
    else {
        
        sdkResetTimer(&hTimer);

        dim3 blocks(imageW / tx + 1, imageH / ty + 1);
        dim3 threads(tx, ty);

        copyHostMemoryToDevice(false);

        copyToDeviceTime = 0.001f * sdkGetTimerValue(&hTimer);
        sdkResetTimer(&hTimer);

        renderImageGPU<<<blocks, threads>>>(d_dst, imageW, imageH, *d_scene);
        checkCudaErrors(hipGetLastError());
        checkCudaErrors(hipDeviceSynchronize());

        calculationTime = 0.001f * sdkGetTimerValue(&hTimer);
        sdkResetTimer(&hTimer);

        copyDeviceToHost();

        copyToHostTime = 0.001f * sdkGetTimerValue(&hTimer);
        //sdkResetTimer(&hTimer);

        float xs, ys;
        xs = ys = 0;

        checkCudaErrors(hipGraphicsMapResources(1, &cuda_pbo_resource, 0));
        size_t num_bytes;
        checkCudaErrors(hipGraphicsResourceGetMappedPointer(
            (void**)&d_dst, &num_bytes, cuda_pbo_resource));

        // Get the pixel scale and offset
        double s = scale / (double)imageW;
        double x = (xs - (double)imageW * 0.5f) * s + xOff;
        double y = (ys - (double)imageH * 0.5f) * s + yOff;

        // Run the mandelbrot generator
        //renderImageGPU(d_dst, imageW, imageH, d_scene);

        // Use the adaptive sampling version when animating.

        //checkCudaErrors(hipMemcpy(d_dst, h_Src, imageW * imageH * sizeof(uchar4),
        //    hipMemcpyHostToDevice));

        checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_pbo_resource, 0));

        if (frameiterator < 15) {
            frameiterator++;
            timeFor60frames += copyToDeviceTime + calculationTime + copyToHostTime;
        }
        else {
            char fps[256];
            sprintf(fps, "<CUDA %s Set> %3.1f fps, %5.8f render time",
                "Sphere Shading", 1.0f / timeFor60frames * 15.0f, timeFor60frames / 15.0f);
            glutSetWindowTitle(fps);
            frameiterator = 0;
            timeFor60frames = 0;
        }

        //printf("GPU = %1.5f, %1.5f, %1.5f\n", copyToDeviceTime, calculationTime, copyToHostTime);
    }
}

static void display(void)
{
    sdkStartTimer(&hTimer);

    // render the Mandelbrot image
    renderImage(g_runcpu);

    // load texture from PBO
    //  glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, gl_PBO);
    glBindTexture(GL_TEXTURE_2D, gl_Tex);
    glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, imageW, imageH, GL_RGBA,
        GL_UNSIGNED_BYTE, BUFFER_DATA(0));
    //  glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, 0);

    // fragment program is required to display floating point texture
    glBindProgramARB(GL_FRAGMENT_PROGRAM_ARB, gl_Shader);
    glEnable(GL_FRAGMENT_PROGRAM_ARB);
    glDisable(GL_DEPTH_TEST);

    glBegin(GL_QUADS);
    glTexCoord2f(0.0f, 0.0f);
    glVertex2f(0.0f, 0.0f);
    glTexCoord2f(1.0f, 0.0f);
    glVertex2f(1.0f, 0.0f);
    glTexCoord2f(1.0f, 1.0f);
    glVertex2f(1.0f, 1.0f);
    glTexCoord2f(0.0f, 1.0f);
    glVertex2f(0.0f, 1.0f);
    glEnd();

    glBindTexture(GL_TEXTURE_2D, 0);
    glDisable(GL_FRAGMENT_PROGRAM_ARB);

    sdkStopTimer(&hTimer);
    glutSwapBuffers();

    lastFrameTime = currentFrameTime;
    currentFrameTime = sdkGetTimerValue(&globalTimer);
    deltaTime = currentFrameTime - lastFrameTime;

    //printf("Width: % d, Height: %d, FPS: %3.1f\n", imageW, imageH, framesPerSecond);
    //printf("pos: %3.1f, %3.1f, %3.1f, right: %3.1f, %3.1f, %3.1f, up: %3.1f, %3.1f, %3.1f \n", 
    //    viewer.getPos().x(), viewer.getPos().y(), viewer.getPos().z(),
    //    viewer.right.x(), viewer.right.y(), viewer.right.z(),
    //    viewer.up.x(), viewer.up.y(), viewer.up.z());

    computeFPS();
}

void timerEvent(int value) {
    if (glutGetWindow()) {
        glutPostRedisplay();
        glutTimerFunc(REFRESH_DELAY, timerEvent, 0);
    }
}

static void idle(void)
{
    glutPostRedisplay();
}

// DATA INITIALIZATION

void initGL(int* argc, char** argv) {
    printf("Initializing GLUT...\n");
    glutInit(argc, argv);

    glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
    glutInitWindowSize(imageW, imageH);
    glutInitWindowPosition(0, 0);
    glutCreateWindow(argv[0]);

    glutDisplayFunc(display);
    glutKeyboardFunc(keyboard);
    glutMouseFunc(clickFunc);
    glutMotionFunc(motionFunc);
    glutReshapeFunc(reshape);
    glutCloseFunc(cleanup);
    glutTimerFunc(REFRESH_DELAY, timerEvent, 0);
    //initMenus();

    if (!isGLVersionSupported(1, 5) ||
        !areGLExtensionsSupported(
            "GL_ARB_vertex_buffer_object GL_ARB_pixel_buffer_object")) {
        fprintf(stderr, "Error: failed to get minimal extensions for demo\n");
        fprintf(stderr, "This sample requires:\n");
        fprintf(stderr, "  OpenGL version 1.5\n");
        fprintf(stderr, "  GL_ARB_vertex_buffer_object\n");
        fprintf(stderr, "  GL_ARB_pixel_buffer_object\n");
        exit(EXIT_SUCCESS);
    }

    printf("OpenGL window created.\n");
}

void initData(int argc, char** argv) {
    // check for hardware double precision support
    int dev = 0;
    dev = findCudaDevice(argc, (const char**)argv);

    hipDeviceProp_t deviceProp;
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, dev));
    version = deviceProp.major * 10 + deviceProp.minor;

    numSMs = deviceProp.multiProcessorCount;

    // initialize some of the arguments
    if (checkCmdLineFlag(argc, (const char**)argv, "xOff")) {
        xOff = getCmdLineArgumentFloat(argc, (const char**)argv, "xOff");
    }

    if (checkCmdLineFlag(argc, (const char**)argv, "yOff")) {
        yOff = getCmdLineArgumentFloat(argc, (const char**)argv, "yOff");
    }

    if (checkCmdLineFlag(argc, (const char**)argv, "scale")) {
        scale = getCmdLineArgumentFloat(argc, (const char**)argv, "xOff");
    }

    printf("Data initialization done.\n");
}

void initPositions(s_positions& positions, int count) {
    positions.count = count;
    positions.x = new float[count];
    positions.y = new float[count];
    positions.z = new float[count];
    positions.angle = new float[count];
}
void initPositionsCUDA(s_positions& positions, int count) {
    checkCudaErrors(hipMalloc((void**)&positions.x, count * sizeof(float)));
    checkCudaErrors(hipMalloc((void**)&positions.y, count * sizeof(float)));
    checkCudaErrors(hipMalloc((void**)&positions.z, count * sizeof(float)));
    checkCudaErrors(hipMalloc((void**)&positions.angle, count * sizeof(float)));
}
void initColors(s_colors& colors, int count) {
    colors.count = count;
    colors.x = new float[count];
    colors.y = new float[count];
    colors.z = new float[count];
}
void initColorsCUDA(s_colors& colors, int count) {
    checkCudaErrors(hipMalloc((void**)&colors.x, count * sizeof(float)));
    checkCudaErrors(hipMalloc((void**)&colors.y, count * sizeof(float)));
    checkCudaErrors(hipMalloc((void**)&colors.z, count * sizeof(float)));
}

void initSpheres(s_scene& scene, int sphere_count, bool useCuda) {

    if (useCuda == true) {
        initPositionsCUDA(scene.spheres.center, sphere_count);
        initColorsCUDA(scene.spheres.colors, sphere_count);

        checkCudaErrors(hipMalloc((void**)&scene.spheres.radius, sizeof(float) * sphere_count));
        checkCudaErrors(hipMalloc((void**)&scene.spheres.ka, sizeof(float) * sphere_count));
        checkCudaErrors(hipMalloc((void**)&scene.spheres.kd, sizeof(float) * sphere_count));
        checkCudaErrors(hipMalloc((void**)&scene.spheres.ks, sizeof(float) * sphere_count));
    }
	else {
        scene.spheres.count = sphere_count;
		initPositions(scene.spheres.center, sphere_count);
		initColors(scene.spheres.colors, sphere_count);

        scene.spheres.radius = new float[sphere_count];
        scene.spheres.ka = new float[sphere_count];
        scene.spheres.kd = new float[sphere_count];
        scene.spheres.ks = new float[sphere_count];
	}
}
void initLights(s_scene& scene, int light_count, bool useCuda) {

    if (useCuda == true) {
        scene.lights.count = light_count;
        initPositionsCUDA(scene.lights.center, light_count);
        checkCudaErrors(hipMalloc((void**)&scene.lights.colors.x, light_count * sizeof(float)));
        checkCudaErrors(hipMalloc((void**)&scene.lights.colors.y, light_count * sizeof(float)));
        checkCudaErrors(hipMalloc((void**)&scene.lights.colors.z, light_count * sizeof(float)));
    }
    else {
        scene.lights.count = light_count;
        initPositions(scene.lights.center, light_count);
        initColors(scene.lights.colors, light_count);
    }
}

void initCamera(s_camera& camera, bool useCuda) {
	camera.position.x = 0.0f;
	camera.position.y = 0.0f;
	camera.position.z = 1.0f;

	set_resolution(h_scene->camera, imageW, imageH);
	look_at(h_scene->camera, make_float3(0, 0, -1));
}

void initScene() {
    h_scene = new s_scene;
    checkCudaErrors(hipMalloc((void**)&d_scene, sizeof(s_scene)));
    // NIE DZIAŁA, PONIEWAŻ PRÓBUJESZ PRZYPISYWAĆ COŚ DO DANYCH ZAALOKOWANYCH NA GPU
    initSpheres(*h_scene, SPHERE_COUNT, false);
    initLights(*h_scene, LIGHT_COUNT, false);

    printf("CPU Spheres initialized\n");

    initSpheres(d_scene_allocationhelper, SPHERE_COUNT, true);
    initLights(d_scene_allocationhelper, LIGHT_COUNT, true);

    printf("GPU Spheres initialized\n");

    initCamera(h_scene->camera, false);

    SceneRandom(h_scene, SPHERE_COUNT, LIGHT_COUNT);

    copyHostMemoryToDevice(true);

    printf("Scene initialized\n");
}

int main(int argc, char* argv[])
{
    initData(argc, argv);
    initScene();

    initGL(&argc, argv);
    initOpenGLBuffers(imageW, imageH);

    sdkCreateTimer(&hTimer);
    sdkStartTimer(&hTimer);

    sdkCreateTimer(&globalTimer);
    sdkStartTimer(&globalTimer);

    glutMainLoop();

    return 0;
}