#include "hip/hip_runtime.h"
﻿
#include <stdio.h>

#include "Dependencies/GL/glew.h"
#include "Dependencies/GL/freeglut.h"

#include "hip/hip_runtime.h"
#include ""
#include "cuda_gl_interop.h"

#include "Dependencies/Helpers/hip/hip_runtime_api.h"
#include "Dependencies/Helpers/helper_gl.h"
#include "Dependencies/Helpers/helper_timer.h"

#include "SphereShadingCPU.h"

#define REFRESH_DELAY 10  // ms

// OpenGL PBO and texture "names"
GLuint gl_PBO, gl_Tex, gl_Shader;
struct hipGraphicsResource* cuda_pbo_resource;  // handles OpenGL-CUDA exchange

// Source image on the host side
uchar4* h_Src = 0;

// Destination image on the GPU side
uchar4* d_dst = NULL;

// Check whether frames should be rendered by CPU or GPU
bool g_runcpu = true;

// Original image width and height
int imageW = 800, imageH = 600;

// Starting position and scale
double xOff = -0.5;
double yOff = 0.0;
double scale = 3.2;

// Starting stationary position and scale motion
double xdOff = 0.0;
double ydOff = 0.0;
double dscale = 1.0;

// Starting color multipliers and random seed
int colorSeed = 0;
uchar4 colors;

int numSMs = 0;   // number of multiprocessors
int version = 1;  // Compute Capability

hittable_list sphere_list;
lights_list lights;
camera viewer;

vec3 moveLEFT = vec3(1, 0, 0);
vec3 moveBACK = vec3(0, 0, 1);
bool cameraRotationMode = false;
int ox, oy;
float angle_x = 0, angle_y = 0;
float start_angle_x, start_angle_y;

static int DisplaySize() {
    return imageW * imageH;
}

// Timer ID
StopWatchInterface* hTimer = NULL;
StopWatchInterface* globalTimer = NULL;

int frameCount, fpsCount, fpsLimit = 15;
float deltaTime = 0;
float lastFrameTime = 0;
float currentFrameTime = 0;
float framesPerSecond;

#ifndef MAX
#define MAX(a, b) ((a > b) ? a : b)
#endif
#define BUFFER_DATA(i) ((char *)0 + i)

void computeFPS() {
    frameCount++;
    fpsCount++;
    if (fpsCount == fpsLimit) {
        char fps[256];
        float ifps = 1.f / (sdkGetAverageTimerValue(&hTimer) / 1000.f);
        framesPerSecond = ifps;
        sprintf(fps, "<CUDA %s Set> %3.1f fps, %5.8f render time",
            "Sphere Shading", ifps, deltaTime * 0.001f);
        glutSetWindowTitle(fps);
        fpsCount = 0;

        fpsLimit = (int)MAX(1.f, (float)ifps);
        sdkResetTimer(&hTimer);
    }
}

// gl_Shader for displaying floating-point texture
static const char* shader_code =
"!!ARBfp1.0\n"
"TEX result.color, fragment.texcoord, texture[0], 2D; \n"
"END";
GLuint compileASMShader(GLenum program_type, const char* code) {
    GLuint program_id;
    glGenProgramsARB(1, &program_id);
    glBindProgramARB(program_type, program_id);
    glProgramStringARB(program_type, GL_PROGRAM_FORMAT_ASCII_ARB,
        (GLsizei)strlen(code), (GLubyte*)code);

    GLint error_pos;
    glGetIntegerv(GL_PROGRAM_ERROR_POSITION_ARB, &error_pos);

    if (error_pos != -1) {
        const GLubyte* error_string;
        error_string = glGetString(GL_PROGRAM_ERROR_STRING_ARB);
        fprintf(stderr, "Program error at position: %d\n%s\n", (int)error_pos,
            error_string);
        return 0;
    }

    return program_id;
}

void initOpenGLBuffers(int w, int h) {
    // delete old buffers
    if (h_Src) {
        free(h_Src);
        h_Src = 0;
    }

    if (gl_Tex) {
        glDeleteTextures(1, &gl_Tex);
        gl_Tex = 0;
    }

    if (gl_PBO) {
        // DEPRECATED: checkCudaErrors(cudaGLUnregisterBufferObject(gl_PBO));
        hipGraphicsUnregisterResource(cuda_pbo_resource);
        glDeleteBuffers(1, &gl_PBO);
        gl_PBO = 0;
    }

    // allocate new buffers
    h_Src = (uchar4*)malloc(w * h * 4);

    printf("Creating GL texture...\n");
    glEnable(GL_TEXTURE_2D);
    glGenTextures(1, &gl_Tex);
    glBindTexture(GL_TEXTURE_2D, gl_Tex);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA8, w, h, 0, GL_RGBA, GL_UNSIGNED_BYTE,
        h_Src);
    printf("Texture created.\n");

    printf("Creating PBO...\n");
    glGenBuffers(1, &gl_PBO);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, gl_PBO);
    glBufferData(GL_PIXEL_UNPACK_BUFFER_ARB, w * h * 4, h_Src, GL_STREAM_COPY);
    // While a PBO is registered to CUDA, it can't be used
    // as the destination for OpenGL drawing calls.
    // But in our particular case OpenGL is only used
    // to display the content of the PBO, specified by CUDA kernels,
    // so we need to register/unregister it only once.

    // DEPRECATED: checkCudaErrors( cudaGLRegisterBufferObject(gl_PBO) );
    checkCudaErrors(hipGraphicsGLRegisterBuffer(
        &cuda_pbo_resource, gl_PBO, cudaGraphicsMapFlagsWriteDiscard));
    printf("PBO created.\n");

    checkCudaErrors(hipMalloc(&d_dst, (w * h * 4) * sizeof(uchar4)));

    // load shader program
    gl_Shader = compileASMShader(GL_FRAGMENT_PROGRAM_ARB, shader_code);
}

static void keyboard(unsigned char key, int /*x*/, int /*y*/) {
	switch (key) {
	case 'w':
		viewer.move(-deltaTime * viewer.speed * moveBACK);
		break;
	case 's':
		viewer.move(deltaTime * viewer.speed * moveBACK);
		break;
	case 'a':
		viewer.move(deltaTime * viewer.speed * moveLEFT);
		break;
	case 'd':
		viewer.move(-deltaTime * viewer.speed * moveLEFT);
		break;
	}

    glutPostRedisplay();
}

void clickFunc(int button, int state, int x, int y) {
    //if (button == GLUT_LEFT_BUTTON) {
    //    if (state == GLUT_DOWN) {
    //        cameraRotationMode = true;
    //        printf("leftclick\n");
    //    }
    //    else {
    //        cameraRotationMode = false;
    //        printf("rightclick\n");
    //    }
    //}

    if (state == GLUT_DOWN) {
        ox = x;
        oy = y;
        start_angle_x = angle_x;
        start_angle_y = angle_y;
    }

    glutPostRedisplay();
}

void motionFunc(int x, int y) {
    
    angle_x = start_angle_x + ((float)(x - ox) / 300.0f);
    angle_y = start_angle_y + ((float)(y - oy) / 300.0f);

    float yaw = deltaTime * viewer.speed * angle_x;
    float pitch = deltaTime * viewer.speed * angle_y;

    //if (pitch > 89.0f) {
    //    pitch = 89.0f;
    //}
    //else if (pitch < -89.0f) {
    //    pitch = -89.0f;
    //}

    //viewer.forward.

    vec3 forward = viewer.forward;

    vec3 newForward = vec3(
        forward.x() - sin(yaw) * cos(pitch),
        forward.x() - sin(pitch),
        forward.x() - cos(yaw) * cos(pitch)
    );

    vec3 right = viewer.right;

    vec3 newRight = vec3(
        right.x() - cos(yaw),
        right.y(),
        right.z() + sin(yaw)
    );

    viewer.setForward(unit_vector(newForward));
    viewer.setRight(unit_vector(newRight));
    viewer.setUp(unit_vector(cross(newForward, newRight)));

    //printf("new UP %3.1f, %3.1f, %3.1f\n", viewer.front.x(), viewer.front.y(), viewer.front.z());

    ox = x;
    oy = y;

    glutPostRedisplay();
}

static void reshape(int w, int h) {
    glViewport(0, 0, w, h);

    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();

    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    glOrtho(0.0, 1.0, 0.0, 1.0, 0.0, 1.0);

    if (w != 0 && h != 0)  // Do not call when window is minimized that is when
                           // width && height == 0
        initOpenGLBuffers(w, h);

    imageW = w;
    imageH = h;

    glutPostRedisplay();
}

static void cleanup() {
    if (h_Src) {
        free(h_Src);
        h_Src = 0;
    }

    sdkStopTimer(&hTimer);
    sdkDeleteTimer(&hTimer);

    sdkStopTimer(&globalTimer);
    sdkDeleteTimer(&globalTimer);

    // DEPRECATED: checkCudaErrors(cudaGLUnregisterBufferObject(gl_PBO));
    checkCudaErrors(hipGraphicsUnregisterResource(cuda_pbo_resource));
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, 0);

    glDeleteBuffers(1, &gl_PBO);
    glDeleteTextures(1, &gl_Tex);
    glDeleteProgramsARB(1, &gl_Shader);

    hipFree(d_dst);
}

void renderImage(bool runcpu) {
    if (runcpu) {
        //int startPass = pass;
        float xs, ys;
        xs = ys = 0;
		sdkResetTimer(&hTimer);

		checkCudaErrors(hipGraphicsMapResources(1, &cuda_pbo_resource, 0));
		size_t num_bytes;
		checkCudaErrors(hipGraphicsResourceGetMappedPointer(
			(void**)&d_dst, &num_bytes, cuda_pbo_resource));

		// Get the pixel scale and offset
		double s = scale / (double)imageW;
		double x = (xs - (double)imageW * 0.5f) * s + xOff;
		double y = (ys - (double)imageH * 0.5f) * s + yOff;

        // Run the mandelbrot generator
        renderImageCPU(h_Src, imageW, imageH, sphere_list, lights, viewer);
        
        // Use the adaptive sampling version when animating.

        checkCudaErrors(hipMemcpy(d_dst, h_Src, imageW * imageH * sizeof(uchar4),
            hipMemcpyHostToDevice));

        checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_pbo_resource, 0));

        //printf("CPU = %5.8f\n", 0.001f * sdkGetTimerValue(&hTimer));
    }
}

static void display(void)
{
    sdkStartTimer(&hTimer);

    // render the Mandelbrot image
    renderImage(g_runcpu);

    // load texture from PBO
    //  glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, gl_PBO);
    glBindTexture(GL_TEXTURE_2D, gl_Tex);
    glTexSubImage2D(GL_TEXTURE_2D, 0, 0, 0, imageW, imageH, GL_RGBA,
        GL_UNSIGNED_BYTE, BUFFER_DATA(0));
    //  glBindBuffer(GL_PIXEL_UNPACK_BUFFER_ARB, 0);

    // fragment program is required to display floating point texture
    glBindProgramARB(GL_FRAGMENT_PROGRAM_ARB, gl_Shader);
    glEnable(GL_FRAGMENT_PROGRAM_ARB);
    glDisable(GL_DEPTH_TEST);

    glBegin(GL_QUADS);
    glTexCoord2f(0.0f, 0.0f);
    glVertex2f(0.0f, 0.0f);
    glTexCoord2f(1.0f, 0.0f);
    glVertex2f(1.0f, 0.0f);
    glTexCoord2f(1.0f, 1.0f);
    glVertex2f(1.0f, 1.0f);
    glTexCoord2f(0.0f, 1.0f);
    glVertex2f(0.0f, 1.0f);
    glEnd();

    glBindTexture(GL_TEXTURE_2D, 0);
    glDisable(GL_FRAGMENT_PROGRAM_ARB);

    sdkStopTimer(&hTimer);
    glutSwapBuffers();

    lastFrameTime = currentFrameTime;
    currentFrameTime = sdkGetTimerValue(&globalTimer);
    deltaTime = currentFrameTime - lastFrameTime;

    //printf("Width: % d, Height: %d, FPS: %3.1f\n", imageW, imageH, framesPerSecond);
    printf("pos: %3.1f, %3.1f, %3.1f, right: %3.1f, %3.1f, %3.1f, up: %3.1f, %3.1f, %3.1f \n", 
        viewer.getPos().x(), viewer.getPos().y(), viewer.getPos().z(),
        viewer.right.x(), viewer.right.y(), viewer.right.z(),
        viewer.up.x(), viewer.up.y(), viewer.up.z());


    computeFPS();
}

void timerEvent(int value) {
    if (glutGetWindow()) {
        glutPostRedisplay();
        glutTimerFunc(REFRESH_DELAY, timerEvent, 0);
    }
}

static void idle(void)
{
    glutPostRedisplay();
}

void initGL(int* argc, char** argv) {
    printf("Initializing GLUT...\n");
    glutInit(argc, argv);

    glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
    glutInitWindowSize(imageW, imageH);
    glutInitWindowPosition(0, 0);
    glutCreateWindow(argv[0]);

    glutDisplayFunc(display);
    glutKeyboardFunc(keyboard);
    glutMouseFunc(clickFunc);
    glutMotionFunc(motionFunc);
    glutReshapeFunc(reshape);
    glutCloseFunc(cleanup);
    glutTimerFunc(REFRESH_DELAY, timerEvent, 0);
    //initMenus();

    if (!isGLVersionSupported(1, 5) ||
        !areGLExtensionsSupported(
            "GL_ARB_vertex_buffer_object GL_ARB_pixel_buffer_object")) {
        fprintf(stderr, "Error: failed to get minimal extensions for demo\n");
        fprintf(stderr, "This sample requires:\n");
        fprintf(stderr, "  OpenGL version 1.5\n");
        fprintf(stderr, "  GL_ARB_vertex_buffer_object\n");
        fprintf(stderr, "  GL_ARB_pixel_buffer_object\n");
        exit(EXIT_SUCCESS);
    }

    printf("OpenGL window created.\n");
}

void initData(int argc, char** argv) {
    // check for hardware double precision support
    int dev = 0;
    dev = findCudaDevice(argc, (const char**)argv);

    hipDeviceProp_t deviceProp;
    checkCudaErrors(hipGetDeviceProperties(&deviceProp, dev));
    version = deviceProp.major * 10 + deviceProp.minor;

    numSMs = deviceProp.multiProcessorCount;

    // initialize some of the arguments
    if (checkCmdLineFlag(argc, (const char**)argv, "xOff")) {
        xOff = getCmdLineArgumentFloat(argc, (const char**)argv, "xOff");
    }

    if (checkCmdLineFlag(argc, (const char**)argv, "yOff")) {
        yOff = getCmdLineArgumentFloat(argc, (const char**)argv, "yOff");
    }

    if (checkCmdLineFlag(argc, (const char**)argv, "scale")) {
        scale = getCmdLineArgumentFloat(argc, (const char**)argv, "xOff");
    }

    printf("Data initialization done.\n");
}

void initScene() {
    sphere_list.add(make_shared<sphere>(point3(0, 0, -1), 0.5f, color(1, 1, 1)));
    sphere_list.add(make_shared<sphere>(point3(0, -100.5, -1), 100, color(1, 1, 1)));
    //lights.add(make_shared<light>(point3(-1, 0, 0), color(0.0f, 0.0f, 2.0f)));
    lights.add(make_shared<light>(point3(0, 1, 0), color(2.0f, 1.2f, 0.0f)));
    //lights.add(make_shared<light>(point3(0, 0, -1), color(0.0f, 1.5f, 0.0f)));

    viewer = camera(point3(0, 0, 0), vec3(1, 0, 0), vec3(0, 0, 1));
}

int main(int argc, char* argv[])
{
    initData(argc, argv);
    initScene();

    initGL(&argc, argv);
    initOpenGLBuffers(imageW, imageH);

    sdkCreateTimer(&hTimer);
    sdkStartTimer(&hTimer);

    sdkCreateTimer(&globalTimer);
    sdkStartTimer(&globalTimer);

    glutMainLoop();

    return 0;
}